#include "hip/hip_runtime.h"
// Copyright (c) 2005 - 2015 Marc de Kamps
// All rights reserved.
//
// Redistribution and use in source and binary forms, with or without modification, are permitted provided that the following conditions are met:
//
//    * Redistributions of source code must retain the above copyright notice, this list of conditions and the following disclaimer.
//    * Redistributions in binary form must reproduce the above copyright notice, this list of conditions and the following disclaimer in the documentation
//      and/or other materials provided with the distribution.
//    * Neither the name of the copyright holder nor the names of its contributors may be used to endorse or promote products derived from this software
//      without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
// WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY
// DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF
// USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING
// NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
//
//      If you use this software in work leading to a scientific publication, you should include a reference there to
//      the 'currently valid reference', which can be found at http://miind.sourceforge.net

#include <iostream>
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cmath>
#include "CudaEuler.cuh"
#include "CSRAdapter.cuh"

using namespace CudaTwoDLib;

const fptype TOLERANCE = 1e-9;

#define checkCudaErrors(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
  if (code != hipSuccess) {
    fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
    if (abort) exit(code);
  }
}

void CSRAdapter::CalculateMeshGridDerivativeIndexed(const std::vector<inttype>& vecindex,
  const std::vector<fptype>& vecrates, const std::vector<fptype>& vecstays,
  const std::vector<fptype>& vecgoes, const std::vector<inttype>& vecoff1s,
  const std::vector<inttype>& vecoff2s, const std::vector<inttype>& vecworking)
{
  for(inttype m = 0; m < _nr_streams - (_nr_m - _transform_offset); m++)
  {
    //std::cout <<  m << " " << vecindex[m] << " " << vecworking[vecindex[m]] << "\n";
    // be careful to use this block size
    inttype numBlocks = (vecworking[vecindex[m]] + _blockSize - 1)/_blockSize;
    CudaCalculateGridDerivativeIndexed<<<numBlocks,_blockSize,0,_streams[m]>>>(_nr_rows[vecindex[m]],vecrates[m],vecstays[m],vecgoes[m],vecoff1s[m],vecoff2s[m],_dydt,_group._mass,_offsets[vecindex[m]],vecworking[vecindex[m]], _group._grid_index[vecindex[m]]);
  }

  inttype m = _transform_offset;
  for(inttype s =  _nr_streams - (_nr_m - _transform_offset); s < _nr_streams; s++)
  {
    // be careful to use this block size
    inttype numBlocks = (_nr_rows[m] + _blockSize - 1)/_blockSize;
    CudaCalculateDerivative<<<numBlocks,_blockSize,0,_streams[s]>>>(_nr_rows[vecindex[m]],vecrates[m],_dydt,_group._mass,_val[vecindex[m]],_ia[vecindex[m]],_ja[vecindex[m]],_group._map,_offsets[vecindex[m]]);
    m++;
  }

  for (inttype m = 0; m < _nr_streams; m++)
      hipStreamSynchronize(_streams[m]);
}

void CSRAdapter::CalculateMeshGridDerivativeBound(const std::vector<inttype>& vecindex,
  const std::vector<fptype>& vecrates, const std::vector<fptype>& vecstays,
  const std::vector<fptype>& vecgoes, const std::vector<inttype>& vecoff1s,
  const std::vector<inttype>& vecoff2s, const std::vector<inttype>& sxs, const std::vector<inttype>& exs)
{
  for(inttype m = 0; m < _nr_streams - (_nr_m - _transform_offset); m++)
  {
    //std::cout << _offsets[vecindex[m]] << " "<< sxs[m] << " " << exs[m] << "\n";
    // be careful to use this block size
    inttype numBlocks = ((exs[vecindex[m]]-sxs[vecindex[m]]) + _blockSize - 1)/_blockSize;
    CudaCalculateGridDerivativeBound<<<numBlocks,_blockSize,0,_streams[m]>>>(_group._n,vecrates[m],vecstays[m],vecgoes[m],vecoff1s[m],vecoff2s[m],_dydt,_group._mass,_offsets[vecindex[m]],sxs[vecindex[m]],exs[vecindex[m]]);
  }

  inttype m = _transform_offset;
  for(inttype s =  _nr_streams - (_nr_m - _transform_offset); s < _nr_streams; s++)
  {
    // be careful to use this block size
    inttype numBlocks = (_nr_rows[m] + _blockSize - 1)/_blockSize;
    CudaCalculateDerivative<<<numBlocks,_blockSize,0,_streams[s]>>>(_nr_rows[vecindex[m]],vecrates[m],_dydt,_group._mass,_val[vecindex[m]],_ia[vecindex[m]],_ja[vecindex[m]],_group._map,_offsets[vecindex[m]]);
    m++;
  }

  for (inttype m = 0; m < _nr_streams; m++)
      hipStreamSynchronize(_streams[m]);
}

void CSRAdapter::SingleTransformStepIndexed(const std::vector<inttype>& vecworking)
{
  for(inttype m = 0; m < _transform_offset; m++)
  {
      // be careful to use this block size
      inttype numBlocks = (vecworking[m] + _blockSize - 1)/_blockSize;
      CudaSingleTransformStepIndexed<<<numBlocks,_blockSize,0,_streams[m]>>>(vecworking[m],_dydt,_group._mass,_val[m],_ia[m],_ja[m],_group._map,_offsets[m],vecworking[m], _group._grid_index[m]);
  }

  for (inttype m = 0; m < _transform_offset; m++)
      hipStreamSynchronize(_streams[m]);
}

void CSRAdapter::SingleTransformStepBound(const std::vector<inttype>& sxs,const std::vector<inttype>& exs)
{
  for(inttype m = 0; m < _transform_offset; m++)
  {
      // be careful to use this block size
      inttype numBlocks = ((exs[m]-sxs[m]) + _blockSize - 1)/_blockSize;
      CudaSingleTransformStepBound<<<numBlocks,_blockSize,0,_streams[m]>>>(_nr_rows[m],_dydt,_group._mass,_val[m],_ia[m],_ja[m],_group._map,_offsets[m],sxs[m],exs[m]);
  }

  for (inttype m = 0; m < _transform_offset; m++)
      hipStreamSynchronize(_streams[m]);
}

void CSRAdapter::AddDerivativeFullIndexed(const std::vector<inttype>& vecworking)
{
  for(inttype m = 0; m < _transform_offset; m++)
  {
    inttype numBlocks = ((vecworking[m]) + _blockSize - 1)/_blockSize;
    EulerStepIndexed<<<_numBlocks,_blockSize,0,_streams[m]>>>(_dydt,_group._mass,_offsets[m],vecworking[m],_group._grid_index[m]);
  }

  for (inttype m = 0; m < _transform_offset; m++)
      hipStreamSynchronize(_streams[m]);
}

void CSRAdapter::AddDerivativeFullBound(const std::vector<inttype>& sxs, const std::vector<inttype>& exs)
{
  for(inttype m = 0; m < _transform_offset; m++)
  {
    inttype numBlocks = ((exs[m]-sxs[m]) + _blockSize - 1)/_blockSize;
    EulerStepBound<<<_numBlocks,_blockSize,0,_streams[m]>>>(_dydt,_group._mass,sxs[m],exs[m]);
  }

  for (inttype m = 0; m < _transform_offset; m++)
      hipStreamSynchronize(_streams[m]);
}

void CSRAdapter::FillMatrixMaps(const std::vector<TwoDLib::CSRMatrix>& vecmat)
{
   for(inttype m = 0; m < vecmat.size(); m++)
   {
       _nval[m] = vecmat[m].Val().size();
       checkCudaErrors(hipMalloc((fptype**)&_val[m],_nval[m]*sizeof(fptype)));
       // dont't depend on Val() being of fptype
       std::vector<fptype> vecval;
       for (fptype val: vecmat[m].Val())
           vecval.push_back(val);
       checkCudaErrors(hipMemcpy(_val[m],&vecval[0],sizeof(fptype)*_nval[m],hipMemcpyHostToDevice));

       _nia[m] = vecmat[m].Ia().size();
       checkCudaErrors(hipMalloc((inttype**)&_ia[m],_nia[m]*sizeof(inttype)));
       std::vector<inttype> vecia;
       for(inttype ia: vecmat[m].Ia())
           vecia.push_back(ia);
       checkCudaErrors(hipMemcpy(_ia[m],&vecia[0],sizeof(inttype)*_nia[m],hipMemcpyHostToDevice));


       _nja[m] = vecmat[m].Ja().size();
       checkCudaErrors(hipMalloc((inttype**)&_ja[m],_nja[m]*sizeof(inttype)));
       std::vector<inttype> vecja;
       for(inttype ja: vecmat[m].Ja())
           vecja.push_back(ja);
       checkCudaErrors(hipMemcpy(_ja[m],&vecja[0],sizeof(inttype)*_nja[m],hipMemcpyHostToDevice));
   }
}


void CSRAdapter::DeleteMatrixMaps()
{
    for(inttype m = 0; m < _nr_m; m++)
    {
        hipFree(_val[m]);
        hipFree(_ia[m]);
        hipFree(_ja[m]);
    }
}

inttype CSRAdapter::NumberIterations(const CudaOde2DSystemAdapter& group, fptype euler_timestep) const
{
    fptype tstep = group._group.MeshObjects()[0].TimeStep();
    for ( const auto& mesh: group._group.MeshObjects() )
        if (fabs(tstep - mesh.TimeStep()) > TOLERANCE){
           std::cerr << "Not all meshes in this group have the same time step. " <<  tstep << " " << mesh.TimeStep() << " " << tstep - mesh.TimeStep()  << std::endl;
           exit(0);
        }
    inttype  n_steps = static_cast<inttype>(std::round(tstep/euler_timestep));

    return n_steps;
}

void CSRAdapter::InspectMass(inttype i)
{
    std::vector<fptype> hostvec(_group._n);
    checkCudaErrors(hipMemcpy(&hostvec[0],_group._mass,sizeof(fptype)*_group._n,hipMemcpyDeviceToHost));
}

CSRAdapter::CSRAdapter(CudaOde2DSystemAdapter& group, const std::vector<TwoDLib::CSRMatrix>& vecmat,
  inttype transform_offset, inttype nr_connections, fptype euler_timestep):
_group(group),
_euler_timestep(euler_timestep),
_nr_iterations(NumberIterations(group,euler_timestep)),
_nr_m(vecmat.size()),
_nr_streams(nr_connections),
_transform_offset(transform_offset),
_nval(std::vector<inttype>(vecmat.size())),
_val(std::vector<fptype*>(vecmat.size())),
_nia(std::vector<inttype>(vecmat.size())),
_ia(std::vector<inttype*>(vecmat.size())),
_nja(std::vector<inttype>(vecmat.size())),
_ja(std::vector<inttype*>(vecmat.size())),
_offsets(this->Offsets(vecmat)),
_nr_rows(this->NrRows(vecmat)),
_blockSize(256),
_numBlocks( (_group._n + _blockSize - 1) / _blockSize)
{
    this->FillMatrixMaps(vecmat);
    this->FillDerivative();
    this->CreateStreams();
}

CSRAdapter::CSRAdapter(CudaOde2DSystemAdapter& group, const std::vector<TwoDLib::CSRMatrix>& vecmat, fptype euler_timestep):
CSRAdapter(group,vecmat, vecmat.size(),vecmat.size(),euler_timestep)
{
}

CSRAdapter::~CSRAdapter()
{
    this->DeleteMatrixMaps();
    this->DeleteDerivative();
    this->DeleteStreams();
}

void CSRAdapter::CreateStreams()
{
    _streams = (hipStream_t *)malloc(_nr_streams*sizeof(hipStream_t));
    for(int i = 0; i < _nr_streams; i++)
       hipStreamCreate(&_streams[i]);
}

void CSRAdapter::DeleteStreams()
{
   free(_streams);
}

void CSRAdapter::FillDerivative()
{
    checkCudaErrors(hipMalloc((fptype**)&_dydt,_group._n*sizeof(fptype)));
}

void CSRAdapter::DeleteDerivative()
{
    hipFree(_dydt);
}

void CSRAdapter::ClearDerivative()
{
  inttype n=_group._n;
  CudaClearDerivative<<<_numBlocks,_blockSize>>>(n,_dydt,_group._mass);
}

std::vector<inttype> CSRAdapter::NrRows(const std::vector<TwoDLib::CSRMatrix>& vecmat) const
{
	std::vector<inttype> vecret;
	for (inttype m = 0; m < vecmat.size(); m++)
		vecret.push_back(vecmat[m].NrRows());
	return vecret;
}


std::vector<inttype> CSRAdapter::Offsets(const std::vector<TwoDLib::CSRMatrix>& vecmat) const
{
	std::vector<inttype> vecret;
	for (inttype m = 0; m < vecmat.size(); m++)
		vecret.push_back(vecmat[m].Offset());
	return vecret;
}

void CSRAdapter::CalculateDerivative(const std::vector<fptype>& vecrates)
{
    for(inttype m = _transform_offset; m < _nr_m; m++)
    {
        // be careful to use this block size
        inttype numBlocks = (_nr_rows[m] + _blockSize - 1)/_blockSize;
        CudaCalculateDerivative<<<numBlocks,_blockSize,0,_streams[m]>>>(_nr_rows[m],vecrates[m],_dydt,_group._mass,_val[m],_ia[m],_ja[m],_group._map,_offsets[m]);
    }

    for (inttype m = _transform_offset; m < _nr_m; m++)
        hipStreamSynchronize(_streams[m]);
}

void CSRAdapter::CalculateGridDerivative(const std::vector<inttype>& vecindex, const std::vector<fptype>& vecrates, const std::vector<fptype>& vecstays, const std::vector<fptype>& vecgoes, const std::vector<inttype>& vecoff1s, const std::vector<inttype>& vecoff2s)
{
    for(inttype m = 0; m < _nr_streams - (_nr_m - _transform_offset); m++)
    {
        // std::cout << _offsets[vecindex[m]]<< " "<< m << " " << vecindex[m] << " " << vecrates[m] <<"\n";
        // be careful to use this block size
        inttype numBlocks = (_nr_rows[vecindex[m]] + _blockSize - 1)/_blockSize;
        CudaCalculateGridDerivative<<<numBlocks,_blockSize,0,_streams[m]>>>(_nr_rows[vecindex[m]],vecrates[m],vecstays[m],vecgoes[m],vecoff1s[m],vecoff2s[m],_dydt,_group._mass,_offsets[vecindex[m]]);
    }

    for (inttype m = 0; m < _nr_streams - (_nr_m - _transform_offset); m++)
        hipStreamSynchronize(_streams[m]);
}

void CSRAdapter::CalculateMeshGridDerivative(const std::vector<inttype>& vecindex,
  const std::vector<fptype>& vecrates, const std::vector<fptype>& vecstays,
  const std::vector<fptype>& vecgoes, const std::vector<inttype>& vecoff1s,
  const std::vector<inttype>& vecoff2s)
{
  for(inttype m = 0; m < _nr_streams - (_nr_m - _transform_offset); m++)
  {
    // be careful to use this block size
    inttype numBlocks = (_nr_rows[vecindex[m]] + _blockSize - 1)/_blockSize;
    CudaCalculateGridDerivative<<<numBlocks,_blockSize,0,_streams[m]>>>(_nr_rows[vecindex[m]],vecrates[m],vecstays[m],vecgoes[m],vecoff1s[m],vecoff2s[m],_dydt,_group._mass,_offsets[vecindex[m]]);
  }

  inttype m = _transform_offset;
  for(inttype s =  _nr_streams - (_nr_m - _transform_offset); s < _nr_streams; s++)
  {
    // be careful to use this block size
    inttype numBlocks = (_nr_rows[m] + _blockSize - 1)/_blockSize;
    CudaCalculateDerivative<<<numBlocks,_blockSize,0,_streams[s]>>>(_nr_rows[vecindex[m]],vecrates[m],_dydt,_group._mass,_val[vecindex[m]],_ia[vecindex[m]],_ja[vecindex[m]],_group._map,_offsets[vecindex[m]]);
    m++;
  }

  for (inttype m = 0; m < _nr_streams; m++)
      hipStreamSynchronize(_streams[m]);
}

void CSRAdapter::SingleTransformStep()
{
  for(inttype m = 0; m < _transform_offset; m++)
  {
      // be careful to use this block size
      inttype numBlocks = (_nr_rows[m] + _blockSize - 1)/_blockSize;
      CudaSingleTransformStep<<<numBlocks,_blockSize,0,_streams[m]>>>(_nr_rows[m],_dydt,_group._mass,_val[m],_ia[m],_ja[m],_group._map,_offsets[m]);
  }

  for (inttype m = 0; m < _transform_offset; m++)
      hipStreamSynchronize(_streams[m]);
}

void CSRAdapter::AddDerivative()
{
  EulerStep<<<_numBlocks,_blockSize>>>(_group._n,_dydt,_group._mass,_euler_timestep);
}

void CSRAdapter::AddDerivativeFull()
{
  EulerStep<<<_numBlocks,_blockSize>>>(_group._n,_dydt,_group._mass, 1.0);
}
